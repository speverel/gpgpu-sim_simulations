#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include <stdlib.h>
#include <string>  

// includes CUDA
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 128
#define NUM_OF_BLOCKS 80

// Variables

__constant__ unsigned ConstArray1[THREADS_PER_BLOCK*NUM_OF_BLOCKS];
unsigned* h_Value;
unsigned* d_Value;


// Functions


////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions




// Device code
__global__ void PowerKernal(unsigned* Value, unsigned* const1, unsigned long long iterations)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // unsigned loadAddr = A+ i;
    // unsigned storeAddr = B+ i;
    unsigned load_value;
  unsigned sum_value = 0;
  
  #pragma unroll 100

    for(unsigned long long k=0; k<iterations;k++) {
      //load_value+=ConstArray1[i];
      __asm volatile(
        "ld.const.u32 %0, [%1];" 
        : "=r"(load_value) : "l"(&const1[i]) : "memory");
      __asm volatile("add.u32 %0, %0, %1;" : "+r"(sum_value) : "r"(load_value));
      // __asm volatile(
      //  "st.global.wb.u32 [%0], %1;"
      //  : : "l"((unsigned long)(B+i)), "r"(load_value) 
      // );

    }
    *Value = sum_value;
    __syncthreads();
}


// Host code

int main(int argc, char** argv) 
{

 unsigned long long iterations;
 char *ptr;
 if (argc != 2){
  fprintf(stderr,"usage: %s #iterations\n",argv[0]);
  exit(1);
 }
 else{
  iterations = strtoull(argv[1], &ptr, 10);
 }

 printf("Power Microbenchmark with %llu iterations\n",iterations);
 int N = THREADS_PER_BLOCK*NUM_OF_BLOCKS;
 unsigned array1[N];
 h_Value = (unsigned *) malloc(sizeof(unsigned));
 for(int i=0; i<N;i++){
	srand((unsigned)time(0));
	array1[i] = rand() / RAND_MAX;
 }


 hipMemcpyToSymbol(HIP_SYMBOL(ConstArray1), array1, sizeof(unsigned) * N );

 checkCudaErrors( hipMalloc((void**)&d_Value, sizeof(unsigned)) );

 hipEvent_t start, stop;
 float elapsedTime = 0;
 checkCudaErrors(hipEventCreate(&start));
 checkCudaErrors(hipEventCreate(&stop));
 dim3 dimGrid(NUM_OF_BLOCKS,1);
 dim3 dimBlock(THREADS_PER_BLOCK,1);

 checkCudaErrors(hipEventRecord(start));
 PowerKernal<<<dimGrid,dimBlock>>>(d_Value, ConstArray1, iterations);
 checkCudaErrors(hipEventRecord(stop));

 checkCudaErrors(hipEventSynchronize(stop));
 checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
 printf("gpu execution time = %.2f s\n", elapsedTime/1000);

 getLastCudaError("kernel launch failure");
 hipDeviceSynchronize();

 // Copy result from device memory to host memory
 // h_C contains the result in host memory
 checkCudaErrors( hipMemcpy(h_Value, d_Value, sizeof(unsigned), hipMemcpyDeviceToHost) );

 checkCudaErrors(hipEventDestroy(start));
 checkCudaErrors(hipEventDestroy(stop));

 return 0;
}







