#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include <stdlib.h>


// includes CUDA
#include <hip/hip_runtime.h>

//includes project
#include<repeat.h>

#define THREADS_PER_BLOCK 256
#define NUM_OF_BLOCKS 640


// Variables
unsigned* h_A;
unsigned* h_C;
unsigned* d_A;
unsigned* d_C;


// Functions
void CleanupResources(void);
void RandomInit(unsigned*, int);


////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
     exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
    fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}

// end of CUDA Helper Functions




// Device code
__global__ void PowerKernal(const unsigned* A,unsigned* C, int iterations)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    unsigned I1=A[i];
    #pragma unroll 1
    //Excessive Logical Unit access
    for(unsigned k=0; k<iterations;k++) {
    // BLOCK-0 (For instruction size of 16 bytes)
    	__asm volatile (	
    	"B0: bra.uni B1;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B1: bra.uni B2;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")
    			
    	"B2: bra.uni B3;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B3: bra.uni B4;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B4: bra.uni B5;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B5: bra.uni B6;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B6: bra.uni B7;\n\t" 
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B7: bra.uni B8;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B8: bra.uni B9;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B9: bra.uni B10;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B10: bra.uni B11;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B11: bra.uni B12;\n\t" 
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B12: bra.uni B13;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B13: bra.uni B14;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B14: bra.uni B15;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B15: bra.uni B16;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B16: bra.uni B17;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B17: bra.uni B18;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B18: bra.uni B19;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B19: bra.uni B20;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B20: bra.uni B21;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B21: bra.uni B22;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B22: bra.uni B23;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B23: bra.uni B24;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B24: bra.uni B25;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B25: bra.uni B26;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B26: bra.uni B27;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B27: bra.uni B28;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B28: bra.uni B29;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B29: bra.uni B30;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B30: bra.uni B31;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")
    	
        "B31: bra.uni B32;\n\t"
        repeat8191("add.u32   %0, %1, 1;\n\t")

        "B32: bra.uni B33;\n\t"
        repeat8191("add.u32   %0, %1, 1;\n\t")

        "B33: bra.uni B34;\n\t"
        repeat8191("add.u32   %0, %1, 1;\n\t")

    	"B34: bra.uni LOOP;\n\t"
    	repeat8191("add.u32   %0, %1, 1;\n\t")
        
    	"LOOP:"
    	: "=r"(I1) : "r"(I1));
    	
    }
    C[i]=I1;
    __syncthreads();

}

// Host code

int main(int argc, char** argv) 
{

  int iterations;
  if (argc != 2){
    fprintf(stderr,"usage: %s #iterations\n",argv[0]);
    exit(1);
  }
  else{
    iterations = atoi(argv[1]);
  }

  printf("Power Microbenchmark with %d iterations\n",iterations);
 int N = THREADS_PER_BLOCK*NUM_OF_BLOCKS;
 size_t size = N * sizeof(unsigned);
 // Allocate input vectors h_A and h_B in host memory
 h_A = (unsigned*)malloc(size);
 if (h_A == 0) CleanupResources();
 h_C = (unsigned*)malloc(size);
 if (h_C == 0) CleanupResources();

 // Initialize input vectors
 RandomInit(h_A, N);

 // Allocate vectors in device memory
 checkCudaErrors( hipMalloc((void**)&d_A, size) );
 checkCudaErrors( hipMalloc((void**)&d_C, size) );

 // Copy vectors from host memory to device memory
 checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );

 hipEvent_t start, stop;
 float elapsedTime = 0;
 checkCudaErrors(hipEventCreate(&start));
 checkCudaErrors(hipEventCreate(&stop));
 dim3 dimGrid(NUM_OF_BLOCKS,1);
 dim3 dimBlock(THREADS_PER_BLOCK,1);

 checkCudaErrors(hipEventRecord(start));
 PowerKernal<<<dimGrid,dimBlock>>>(d_A,d_C, iterations);
 checkCudaErrors(hipEventRecord(stop));

 checkCudaErrors(hipEventSynchronize(stop));
 checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
 printf("gpu execution time = %.2f s\n", elapsedTime/1000);

 getLastCudaError("kernel launch failure");
 hipDeviceSynchronize();

 // Copy result from device memory to host memory
 // h_C contains the result in host memory
 checkCudaErrors(hipEventDestroy(start));
 checkCudaErrors(hipEventDestroy(stop));
 checkCudaErrors( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
 
 CleanupResources();

 return 0;
}

void CleanupResources(void)
{
  // Free device memory
  if (d_A)
    hipFree(d_A);
  if (d_C)
    hipFree(d_C);

  // Free host memory
  if (h_A)
    free(h_A);
  if (h_C)
    free(h_C);

}

// Allocates an array with random float entries.
void RandomInit(unsigned* data, int n)
{
  for (int i = 0; i < n; ++i){
    srand((unsigned)time(0));  
    data[i] = rand() / RAND_MAX;
  }
}






