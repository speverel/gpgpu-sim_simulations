#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include <stdlib.h>


// includes CUDA
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256
#define NUM_OF_BLOCKS 640

// Variables

__constant__ float ConstArray1[THREADS_PER_BLOCK];
__constant__ float ConstArray2[THREADS_PER_BLOCK];
__constant__ float ConstArray3[THREADS_PER_BLOCK];

float* h_Value;
float* d_Value;

// Functions
void CleanupResources(void);


////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions




// Device code
__global__ void PowerKernal(float* Value, int iterations)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    
    //Do Some Computation
    float Value1;
    float Value2;
    float Value3;
    *Value=1;
    for(unsigned k=0; k<iterations;k++) {
    	Value1=ConstArray1[(i+k)%THREADS_PER_BLOCK];
    	Value2=ConstArray2[(i+k+1)%THREADS_PER_BLOCK];
    	Value3=ConstArray3[(i+k+5)%THREADS_PER_BLOCK];
    	Value2*=Value1;
    	Value3*=Value2;
    	*Value*=Value3;
    }		
     __syncthreads();
    
}


// Host code

int main(int argc, char** argv) 
{
    int iterations;
    if (argc != 2){
        fprintf(stderr,"usage: %s #iterations\n",argv[0]);
        exit(1);
    }
    else{
        iterations = atoi(argv[1]);
    }

  printf("Power Microbenchmark with %d iterations\n",iterations);
 float array1[THREADS_PER_BLOCK];
 h_Value = (float *) malloc(sizeof(float));
 for(int i=0; i<THREADS_PER_BLOCK;i++){
	srand(time(0));
	array1[i] = rand() / RAND_MAX;
 }
 float array2[THREADS_PER_BLOCK];
 for(int i=0; i<THREADS_PER_BLOCK;i++){
	srand(time(0));
	array2[i] = rand() / RAND_MAX;
 }
 float array3[THREADS_PER_BLOCK];
 for(int i=0; i<THREADS_PER_BLOCK;i++){
	srand(time(0));
	array3[i] = rand() / RAND_MAX;
 }

 hipMemcpyToSymbol(HIP_SYMBOL(ConstArray1), array1, sizeof(float) * THREADS_PER_BLOCK );
 hipMemcpyToSymbol(HIP_SYMBOL(ConstArray2), array2, sizeof(float) * THREADS_PER_BLOCK );
 hipMemcpyToSymbol(HIP_SYMBOL(ConstArray3), array3, sizeof(float) * THREADS_PER_BLOCK );
 
 checkCudaErrors( hipMalloc((void**)&d_Value, sizeof(float)) );
 //VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
 dim3 dimGrid(NUM_OF_BLOCKS,1);
 dim3 dimBlock(THREADS_PER_BLOCK,1);

  hipEvent_t start, stop;
  float elapsedTime = 0;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  checkCudaErrors(hipEventRecord(start));
  PowerKernal<<<dimGrid,dimBlock>>>(d_Value, iterations);
  checkCudaErrors(hipEventRecord(stop));
 
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
  printf("gpu execution time = %.2f s\n", elapsedTime/1000);
  getLastCudaError("kernel launch failure");
  hipDeviceSynchronize();
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));
  return 0;
}







