#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
//#include <cutil.h>
// Includes
//#include <stdio.h>

// includes, project
//#include "../include/sdkHelper.h"  // helper for shared functions common to CUDA SDK samples
//#include <shrQATest.h>
//#include <shrUtils.h>

// includes CUDA
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 1024
#define NUM_OF_BLOCKS 640
//#define ITERATIONS 40
//#include "../include/ContAcq-IntClk.h"

// Variables
float* h_A;
float* h_B;
float* h_C;
float* d_A;
float* d_B;
float* d_C;
//bool noprompt = false;
//unsigned int my_timer;

// Functions
void CleanupResources(void);
void RandomInit(float*, int);
//void ParseArguments(int, char**);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions




// Device code
__global__ void PowerKernal1(const float* A, const float* B, float* C, int iterations)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    float Value1=0;
    float Value2=0;
    float Value3=0;
    float Value=0;
    float I1=A[i];
    float I2=B[i];
#pragma unroll 1000
    // Excessive Addition access
    for(int k=0; k<iterations;k++) {
    	Value1=I1+I2;
    	Value3=I1-I2;
    	Value1+=Value2;
    	Value1+=Value2;
    	Value2=Value3-Value1;
    	Value1=Value2+Value3;
    }
    __syncthreads();

    Value=Value1;
    C[i]=Value+Value2;

}

int main(int argc, char** argv)
{
  int iterations;
  unsigned blocks;
  if (argc != 3){
    fprintf(stderr,"usage: %s #iterations #cores\n",argv[0]);
    exit(1);
  }
  else {
    iterations = atoi(argv[1]);
    blocks = atoi(argv[2]);
  }
 
 printf("Power Microbenchmarks with iterations %d\n",iterations);
 int N = THREADS_PER_BLOCK*blocks;
 size_t size = N * sizeof(float);
 // Allocate input vectors h_A and h_B in host memory
 h_A = (float*)malloc(size);
 if (h_A == 0) CleanupResources();
 h_B = (float*)malloc(size);
 if (h_B == 0) CleanupResources();
 h_C = (float*)malloc(size);
 if (h_C == 0) CleanupResources();

 // Initialize input vectors
 RandomInit(h_A, N);
 RandomInit(h_B, N);

 // Allocate vectors in device memory
printf("before\n");
 checkCudaErrors( hipMalloc((void**)&d_A, size) );
 checkCudaErrors( hipMalloc((void**)&d_B, size) );
 checkCudaErrors( hipMalloc((void**)&d_C, size) );
printf("after\n");

 hipEvent_t start, stop;                   
 float elapsedTime = 0;                     
 checkCudaErrors(hipEventCreate(&start));  
 checkCudaErrors(hipEventCreate(&stop));
 
 // Copy vectors from host memory to device memory
 checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
 checkCudaErrors( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );

 //VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
 dim3 dimGrid(blocks,1);
 dim3 dimBlock(THREADS_PER_BLOCK,1);
 dim3 dimGrid2(1,1);
 dim3 dimBlock2(1,1);

 checkCudaErrors(hipEventRecord(start));              
 PowerKernal1<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, iterations);  
 checkCudaErrors(hipEventRecord(stop));               
 
 checkCudaErrors(hipEventSynchronize(stop));           
 checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));  
 printf("execution time = %.2f s\n", elapsedTime/1000);  
 getLastCudaError("kernel launch failure");              
 hipDeviceSynchronize();

/*CUT_SAFE_CALL(cutCreateTimer(&my_timer)); 
TaskHandle taskhandle = LaunchDAQ();
CUT_SAFE_CALL(cutStartTimer(my_timer)); 
printf("execution time = %f\n", cutGetTimerValue(my_timer));



PowerKernal1<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
CUDA_SAFE_CALL( hipDeviceSynchronize() );
printf("execution time = %f\n", cutGetTimerValue(my_timer));


getLastCudaError("kernel launch failure");
CUDA_SAFE_CALL( hipDeviceSynchronize() );
CUT_SAFE_CALL(cutStopTimer(my_timer));
TurnOffDAQ(taskhandle, cutGetTimerValue(my_timer));
printf("execution time = %f\n", cutGetTimerValue(my_timer));
CUT_SAFE_CALL(cutDeleteTimer(my_timer)); 

#ifdef _DEBUG
 checkCudaErrors( hipDeviceSynchronize() );
#endif*/

 // Copy result from device memory to host memory
 // h_C contains the result in host memory
 checkCudaErrors( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
 
 checkCudaErrors(hipEventDestroy(start));
 checkCudaErrors(hipEventDestroy(stop));
 CleanupResources();

 return 0;
}

void CleanupResources(void)
{
  // Free device memory
  if (d_A)
	hipFree(d_A);
  if (d_B)
	hipFree(d_B);
  if (d_C)
	hipFree(d_C);

  // Free host memory
  if (h_A)
	free(h_A);
  if (h_B)
	free(h_B);
  if (h_C)
	free(h_C);

}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
  for (int i = 0; i < n; ++i){ 
	data[i] = rand() / RAND_MAX;
  }
}






