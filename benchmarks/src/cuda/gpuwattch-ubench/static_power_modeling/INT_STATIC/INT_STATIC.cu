#include "hip/hip_runtime.h"
#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 1024
#define DATA_TYPE uint32_t
  
// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}


template <class T>
__global__ void power_microbench(T *data1, T *data2, T *res, int div, unsigned iterations) {

  int gid = blockIdx.x*blockDim.x + threadIdx.x;
  register T s1 = data1[gid];
  register T s2 = data2[gid];
  register T result = 0;
  register T Value1=0;
  register T Value2=0;

  // synchronize all threads
  asm volatile ("bar.sync 0;");

  if((gid%32)<div){
  //ROI
    #pragma unroll 100
    for (unsigned j=0 ; j<iterations ; ++j) {
      asm volatile ("{\t\n"
          "add.u32 %0, %2, %0;\n\t"
          "add.u32 %0, %1, %0;\n\t"
          // "add.u32 %2, %2, %0;\n\t"
          // "mul.lo.u32 %1, %0, %2;\n\t"
          "mad.lo.u32 %2, %2, %2 , %0;\n\t"
          "}" : "+r"(result),"+r"(s1),"+r"(s2)
      );
      // result=s1+s2;
      // Value2=s1-s2;
      // result+=Value1;
      // result*=Value1;
      // Value1=Value2+result;
      // result=Value1+Value2;
    }
  }

  // synchronize all threads
  asm volatile("bar.sync 0;");

  // write data back to memory
  res[gid] = result;
}

int main(int argc, char** argv){
  unsigned iterations;
  int blocks;
  int div;
  if (argc != 4){
    fprintf(stderr,"usage: %s #iterations #cores #ActiveThreadsperWarp\n",argv[0]);
    exit(1);
  }
  else {
    iterations = atoi(argv[1]);
    blocks = atoi(argv[2]);
    div = atoi(argv[3]);
  }
 
 printf("Power Microbenchmarks with iterations %u\n",iterations);
 int total_threads = THREADS_PER_BLOCK*blocks;


DATA_TYPE *data1 = (DATA_TYPE*) malloc(total_threads*sizeof(DATA_TYPE));
DATA_TYPE *data2 = (DATA_TYPE*) malloc(total_threads*sizeof(DATA_TYPE));
DATA_TYPE *res = (DATA_TYPE*) malloc(total_threads*sizeof(DATA_TYPE));

DATA_TYPE *data1_g;
DATA_TYPE *data2_g;
DATA_TYPE *res_g;

for (uint32_t i=0; i<total_threads; i++) {
  srand((unsigned)time(0));  
  data1[i] = (DATA_TYPE) rand() / RAND_MAX;
  srand((unsigned)time(0));
  data2[i] = (DATA_TYPE) rand() / RAND_MAX;
}

gpuErrchk( hipMalloc(&data1_g, total_threads*sizeof(DATA_TYPE)) );
gpuErrchk( hipMalloc(&data2_g, total_threads*sizeof(DATA_TYPE)) );
gpuErrchk( hipMalloc(&res_g, total_threads*sizeof(DATA_TYPE)) );

gpuErrchk( hipMemcpy(data1_g, data1, total_threads*sizeof(DATA_TYPE), hipMemcpyHostToDevice) );
gpuErrchk( hipMemcpy(data2_g, data2, total_threads*sizeof(DATA_TYPE), hipMemcpyHostToDevice) );

power_microbench<DATA_TYPE><<<blocks,THREADS_PER_BLOCK>>>(data1_g, data2_g, res_g, div, iterations);
gpuErrchk( hipPeekAtLastError() );


gpuErrchk( hipMemcpy(res, res_g, total_threads*sizeof(DATA_TYPE), hipMemcpyDeviceToHost) );


hipFree(data1_g);
hipFree(data2_g);
hipFree(res_g);
free(data1);
free(data2);
free(res);

  return 0;
} 