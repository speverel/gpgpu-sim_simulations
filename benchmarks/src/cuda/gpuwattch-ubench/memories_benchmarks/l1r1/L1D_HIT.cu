#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
//#include <cutil.h>
//#include <mgp.h>
// Includes
//#include <stdio.h>
//#include "../include/ContAcq-IntClk.h"

// includes, project
//#include "../include/sdkHelper.h"  // helper for shared functions common to CUDA SDK samples
//#include <shrQATest.h>
//#include <shrUtils.h>

// includes CUDA
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 2048
#define NUM_OF_BLOCKS 80
#define NUM_SM 80
#define LINE_SIZE 	128
#define SETS		4
#define ASSOC		256


// Variables
unsigned* h_A;
unsigned* h_B;
unsigned* h_C;
unsigned* d_A;
unsigned* d_B;
unsigned* d_C;
//bool noprompt = false;
//unsigned int my_timer;

// Functions
void CleanupResources(void);
void RandomInit(unsigned*, int);
//void ParseArguments(int, char**);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions



__global__ void PowerKernal2( unsigned* A, unsigned* B, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // unsigned loadAddr = A+ i;
    // unsigned storeAddr = B+ i;
    unsigned load_value;
	//unsigned sum_value = 0;
	#pragma unroll 100

    for(unsigned k=0; k<N;k++) {
    	__asm volatile(
    		"ld.global.ca.u32 %0, [%1];" 
    		: "=r"(load_value) : "l"((unsigned long)(A+i))
    	);
    	//__asm volatile("add.u32 %0, %0, %1;" : "+r"(sum_value) : "r"(load_value));
    	__asm volatile(
    		"st.global.wb.u32 [%0], %1;"
    		: : "l"((unsigned long)(B+i)), "r"(load_value) 
    	);

    }
    //B[i] = sum_value;
    __syncthreads();

}


int main(int argc, char** argv)
{
 int iterations;
 if(argc!=2) {
   fprintf(stderr,"usage: %s #iterations\n",argv[0]);
   exit(1);
 }
 else {
   iterations = atoi(argv[1]);
 }
 
 printf("Power Microbenchmarks with iterations %d\n",iterations);
 int size_per_sm = (LINE_SIZE*ASSOC*SETS); //131072
 int N = THREADS_PER_BLOCK*NUM_OF_BLOCKS;

 size_t size = size_per_sm*NUM_SM < N * sizeof(int) ? size_per_sm*NUM_OF_BLOCKS : N * sizeof(int);
 // Allocate input vectors h_A and h_B in host memory
 h_A = (unsigned*)malloc(size);
 if (h_A == 0) CleanupResources();
 h_B = (unsigned*)malloc(size);
 if (h_B == 0) CleanupResources();


 // Initialize input vectors
 RandomInit(h_A, N);


 // Allocate vectors in device memory
 checkCudaErrors( hipMalloc((void**)&d_A, size) );
 checkCudaErrors( hipMalloc((void**)&d_B, size) );


 // Copy vectors from host memory to device memory
 checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );


 hipEvent_t start, stop;                   
 float elapsedTime = 0;                     
 checkCudaErrors(hipEventCreate(&start));  
 checkCudaErrors(hipEventCreate(&stop));

 //VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
 dim3 dimGrid(NUM_OF_BLOCKS,1);
 dim3 dimBlock(THREADS_PER_BLOCK,1);
 dim3 dimGrid2(1,1);
 dim3 dimBlock2(1,1);

 checkCudaErrors(hipEventRecord(start));              
 PowerKernal2<<<dimGrid,dimBlock>>>(d_A, d_B,iterations);  
 checkCudaErrors(hipEventRecord(stop));               
 
 checkCudaErrors(hipEventSynchronize(stop));           
 checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));  
 printf("execution time = %.2f s\n", elapsedTime/1000);  
 getLastCudaError("kernel launch failure");              
 hipDeviceSynchronize(); 

/* CUT_SAFE_CALL(cutCreateTimer(&my_timer)); 
 TaskHandle taskhandle = LaunchDAQ();
 CUT_SAFE_CALL(cutStartTimer(my_timer)); 
 printf("execution time = %f\n", cutGetTimerValue(my_timer));

profileKernel("BE_SP_INT_ADD", "PowerKernal2");
for (int i = 0; i < 1000; i++)
{
	PowerKernal2<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
	CUDA_SAFE_CALL( hipDeviceSynchronize() );
}
haltProfiling();
printf("execution time = %f\n", cutGetTimerValue(my_timer));

getLastCudaError("kernel launch failure");
CUDA_SAFE_CALL( hipDeviceSynchronize() );
CUT_SAFE_CALL(cutStopTimer(my_timer));
TurnOffDAQ(taskhandle, cutGetTimerValue(my_timer));
printf("execution time = %f\n", cutGetTimerValue(my_timer));
CUT_SAFE_CALL(cutDeleteTimer(my_timer)); 

#ifdef _DEBUG
 checkCudaErrors( hipDeviceSynchronize() );
#endif*/

 // Copy result from device memory to host memory
 // h_C contains the result in host memory
 checkCudaErrors( hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost) );
  checkCudaErrors(hipEventDestroy(start));
 checkCudaErrors(hipEventDestroy(stop));
 CleanupResources();

 return 0;
}

void CleanupResources(void)
{
  // Free device memory
  if (d_A)
	hipFree(d_A);
  if (d_B)
	hipFree(d_B);

  // Free host memory
  if (h_A)
	free(h_A);
  if (h_B)
	free(h_B);

}

// Allocates an array with random float entries.
void RandomInit(unsigned* data, int n)
{
  for (int i = 0; i < n; ++i){
	srand((unsigned)time(0));  
	data[i] = rand() / RAND_MAX;
  }
}