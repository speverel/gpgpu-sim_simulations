//This code is a modification of L1 cache benchmark from 
//"Dissecting the NVIDIA Volta GPU Architecture via Microbenchmarking": https://arxiv.org/pdf/1804.06826.pdf

//This benchmark stresses the L1 cache

//This code have been tested on Volta V100 architecture

#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define THREADS_NUM 1024  
#define NUM_BLOCKS 160
#define WARP_SIZE 32

// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
        if (code != hipSuccess) {
                fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
                if (abort) exit(code);
        }
}

__global__ void l1_pointers_init(uint64_t *posArray){

  uint32_t tid = blockIdx.x*blockDim.x + threadIdx.x;
  if(tid == 0){
    for(uint32_t blk = 0; blk <NUM_BLOCKS; blk++){
      for (uint32_t i=0; i<(THREADS_NUM-1); i++){
        posArray[(blk*THREADS_NUM)+i] = (uint64_t)(posArray + (blk*THREADS_NUM) + i + 1);
      }

      posArray[((blk+1)*THREADS_NUM)-1] = (uint64_t)(posArray + (blk*THREADS_NUM));
    }
  }
}

__global__ void l1_stress(uint64_t *posArray, uint64_t *dsink, uint64_t iterations){

  // thread index
  uint32_t tid = blockIdx.x*blockDim.x + threadIdx.x;

  if(tid < NUM_BLOCKS*THREADS_NUM){
  	// a register to avoid compiler optimization
  	uint64_t *ptr = posArray + tid;
  	uint64_t ptr1, ptr0;

  	// initialize the thread pointer with the start address of the array
  	// use ca modifier to cache the in L1
  	asm volatile ("{\t\n"
  	  "ld.global.ca.u64 %0, [%1];\n\t"
  	  "}" : "=l"(ptr1) : "l"(ptr) : "memory"
  	);

  	// synchronize all threads
  	asm volatile ("bar.sync 0;");

  	// pointer-chasing iterations times
  	// use ca modifier to cache the load in L1
  	#pragma unroll 100
  	for(uint64_t i=0; i<iterations; ++i) { 
  	  asm volatile ("{\t\n"
  	    "ld.global.ca.u64 %0, [%1];\n\t"
  	    "}" : "=l"(ptr0) : "l"((uint64_t*)ptr1) : "memory"
  	  );
  	  ptr1 = ptr0;    //swap the register for the next load

  	}

  	// write data back to memory
  	dsink[tid] = ptr1;
  }
}

int main(int argc, char** argv){
  uint64_t iterations;
  if (argc != 2){
    fprintf(stderr,"usage: %s #iterations #cores #ActiveThreadsperWarp\n",argv[0]);
    exit(1);
  }
  else {
    iterations = atoll(argv[1]);
  }
  int total_threads = THREADS_NUM*NUM_BLOCKS;
 printf("Power Microbenchmarks with iterations %lu\n",iterations);

  uint64_t *dsink = (uint64_t*) malloc(total_threads*sizeof(uint64_t));
  

  uint64_t *posArray_g;
  uint64_t *dsink_g;
  

  gpuErrchk( hipMalloc(&posArray_g, total_threads*sizeof(uint64_t)) );
  gpuErrchk( hipMalloc(&dsink_g, total_threads*sizeof(uint64_t)) );
  l1_pointers_init<<<1,1>>>(posArray_g);
  
  l1_stress<<<NUM_BLOCKS,THREADS_NUM>>>(posArray_g, dsink_g, iterations);
  gpuErrchk( hipPeekAtLastError() );

  gpuErrchk( hipMemcpy(dsink, dsink_g, total_threads*sizeof(uint64_t), hipMemcpyDeviceToHost) );

  return 0;
} 