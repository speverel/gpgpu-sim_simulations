#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include <stdlib.h>
// includes from project


// includes from CUDA
#include <hip/hip_runtime.h>
//#include <hip/hip_vector_types.h>

#define THREADS_PER_BLOCK 256
#define NUM_OF_BLOCKS 640


// Variables
unsigned* h_A;
unsigned* h_B;
unsigned* h_C;
unsigned* d_A;
unsigned* d_B;
unsigned* d_C;

// Functions
void CleanupResources(void);
void RandomInit(unsigned*, int);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
  fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
   exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
  fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
  exit(-1);
  }
}
// end of CUDA Helper Functions

__global__ void PowerKernal2(const unsigned* A, const unsigned* B, unsigned* C, int iterations)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  //Do Some Computation
  unsigned Value1=1;
  unsigned Value2=A[i];
  unsigned Value3=B[i];
  unsigned Value;
  unsigned I1=A[i];
  unsigned I2=B[i];

  // Excessive INT addition access
  if((i%32)<=7){
    #pragma unroll 100
    for(unsigned k=0; k<iterations;k++) {
      Value2= I1*Value1;
      Value3=I2*Value3;
      Value1*=Value2;
      Value3*=Value1;
      Value2*=Value3;
      Value1*=Value3;
    }
  }
  __syncthreads();

  Value=Value1;
  C[i]=Value;
  __syncthreads();
}

int main(int argc, char** argv) 
{

  int iterations;
  if (argc != 2){
  fprintf(stderr,"usage: %s #iterations\n",argv[0]);
  exit(1);
  }
  else{
    iterations = atoi(argv[1]);
  }

 printf("Power Microbenchmark with %d iterations\n",iterations);
 int N = THREADS_PER_BLOCK*NUM_OF_BLOCKS;
 size_t size = N * sizeof(unsigned);
 // Allocate input vectors h_A and h_B in host memory
 h_A = (unsigned*)malloc(size);
 if (h_A == 0) CleanupResources();
 h_B = (unsigned*)malloc(size);
 if (h_B == 0) CleanupResources();
 h_C = (unsigned*)malloc(size);
 if (h_C == 0) CleanupResources();



 // Initialize input vectors
 RandomInit(h_A, N);
 RandomInit(h_B, N);

 // Allocate vectors in device memory
 printf("before\n");
 checkCudaErrors( hipMalloc((void**)&d_A, size) );
 checkCudaErrors( hipMalloc((void**)&d_B, size) );
 checkCudaErrors( hipMalloc((void**)&d_C, size) );
 printf("after\n");

 hipEvent_t start, stop;
 float elapsedTime = 0;
 checkCudaErrors(hipEventCreate(&start));
 checkCudaErrors(hipEventCreate(&stop));

 // Copy vectors from host memory to device memory
 checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
 checkCudaErrors( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );


 dim3 dimGrid(NUM_OF_BLOCKS,1);
 dim3 dimBlock(THREADS_PER_BLOCK,1);
 dim3 dimGrid2(1,1);
 dim3 dimBlock2(1,1);

 checkCudaErrors(hipEventRecord(start));
 PowerKernal2<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, iterations);
 checkCudaErrors(hipEventRecord(stop));

 checkCudaErrors(hipEventSynchronize(stop));
 checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
 printf("gpu execution time = %.2f s\n", elapsedTime/1000);

 getLastCudaError("kernel launch failure");
 hipDeviceSynchronize();

 // Copy result from device memory to host memory
 // h_C contains the result in host memory
 checkCudaErrors( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
 
 checkCudaErrors(hipEventDestroy(start));
 checkCudaErrors(hipEventDestroy(stop));
 CleanupResources();

 return 0;
}

void CleanupResources(void)
{
  // Free device memory
  if (d_A)
  hipFree(d_A);
  if (d_B)
  hipFree(d_B);
  if (d_C)
  hipFree(d_C);

  // Free host memory
  if (h_A)
  free(h_A);
  if (h_B)
  free(h_B);
  if (h_C)
  free(h_C);

}

// Allocates an array with random unsigned entries.
void RandomInit(unsigned* data, int n)
{
  for (int i = 0; i < n; ++i){
  srand((unsigned)time(0));  
  data[i] = rand() / RAND_MAX;
  }
}