#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include <stdlib.h>


// includes CUDA
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256
#define NUM_OF_BLOCKS 640

// Variables



// Functions

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions




// Device code
__global__ void PowerKernal(int iterations)
{
    int i = threadIdx.x;
    //Do Some Computation
   __device__  __shared__ unsigned I1[THREADS_PER_BLOCK];
   __device__  __shared__ unsigned I2[THREADS_PER_BLOCK];
   __device__ __shared__ float I3[THREADS_PER_BLOCK];
   __device__  __shared__ float I4[THREADS_PER_BLOCK];
	
    I1[i]=i*2;
    I2[i]=i;
    I3[i]=i/2;
    I4[i]=i;

    __syncthreads();

    for(unsigned k=0; k<iterations ;k++) {
        		I1[i]=I2[(i+k)%THREADS_PER_BLOCK];
        		I2[i]=I1[(i+k+1)%THREADS_PER_BLOCK];
    }		
         
    for(unsigned k=0; k<iterations ;k++) {
    			I3[i]=I4[(i+k)%THREADS_PER_BLOCK];
    			I4[i]=I3[(i+k+1)%THREADS_PER_BLOCK];
    } 
    __syncthreads();

}


// Host code

int main(int argc, char** argv) 
{

 int iterations;
 if (argc != 2){
  fprintf(stderr,"usage: %s #iterations\n",argv[0]);
  exit(1);
 }
 else{
  iterations = atoi(argv[1]);
 }

 printf("Power Microbenchmark with %d iterations\n",iterations);


 // Allocate vectors in device memory

 hipEvent_t start, stop;
 float elapsedTime = 0;
 checkCudaErrors(hipEventCreate(&start));
 checkCudaErrors(hipEventCreate(&stop));
  
 dim3 dimGrid(NUM_OF_BLOCKS,1);
 dim3 dimBlock(THREADS_PER_BLOCK,1);

 checkCudaErrors(hipEventRecord(start));
 PowerKernal<<<dimGrid,dimBlock>>>(iterations);

 checkCudaErrors(hipEventRecord(stop));

 checkCudaErrors(hipEventSynchronize(stop));
 checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
 printf("gpu execution time = %.2f s\n", elapsedTime/1000);

 getLastCudaError("kernel launch failure");
 hipDeviceSynchronize();
 checkCudaErrors(hipEventDestroy(start));
 checkCudaErrors(hipEventDestroy(stop));

 return 0;
}








