#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
//#include <cutil.h>
// Includes
//#include <stdio.h>

// includes, project
//#include "../include/sdkHelper.h"  // helper for shared functions common to CUDA SDK samples
//#include <shrQATest.h>
//#include <shrUtils.h>

// includes CUDA
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256
#define NUM_OF_BLOCKS 640
//#define ITERATIONS 40
//#include "../include/ContAcq-IntClk.h"

// Variables
double* h_A;
double* h_B;
double* h_C;
double* d_A;
double* d_B;
double* d_C;
//bool noprompt = false;
//unsigned int my_timer;

// Functions
void CleanupResources(void);
void RandomInit(double*, int);
//void ParseArguments(int, char**);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions




// Device code
__global__ void PowerKernal1(const double* A, const double* B, double* C, int iterations)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    double Value1=0;
    double Value2=0;
    double Value3=0;
    double Value=0;
    double I1=A[i];
    double I2=B[i];

    // Excessive Addition access
    for(unsigned k=0; k<iterations;k++) {
    	Value1=I1+I2;
    	Value3=I1-I2;
    	Value1+=Value2;
    	Value1+=Value2;
    	Value2=Value3-Value1;
    	Value1=Value2+Value3;
    }
    __syncthreads();

    Value=Value1;
    C[i]=Value+Value2;

}

int main(int argc, char** argv)
{
 int iterations;
 if(argc!=2) {
   fprintf(stderr,"usage: %s #iterations\n",argv[0]);
 }
 else {
   iterations = atoi(argv[1]);
 }
 
 printf("Power Microbenchmarks with iterations %d\n",iterations);
 int N = THREADS_PER_BLOCK*NUM_OF_BLOCKS;
 size_t size = N * sizeof(double);
 // Allocate input vectors h_A and h_B in host memory
 h_A = (double*)malloc(size);
 if (h_A == 0) CleanupResources();
 h_B = (double*)malloc(size);
 if (h_B == 0) CleanupResources();
 h_C = (double*)malloc(size);
 if (h_C == 0) CleanupResources();

 // Initialize input vectors
 RandomInit(h_A, N);
 RandomInit(h_B, N);

 // Allocate vectors in device memory
printf("before\n");
 checkCudaErrors( hipMalloc((void**)&d_A, size) );
 checkCudaErrors( hipMalloc((void**)&d_B, size) );
 checkCudaErrors( hipMalloc((void**)&d_C, size) );
printf("after\n");

 hipEvent_t start, stop;                   
 float elapsedTime = 0;                     
 checkCudaErrors(hipEventCreate(&start));  
 checkCudaErrors(hipEventCreate(&stop));
 
 // Copy vectors from host memory to device memory
 checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
 checkCudaErrors( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );

 //VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
 dim3 dimGrid(NUM_OF_BLOCKS,1);
 dim3 dimBlock(THREADS_PER_BLOCK,1);
 dim3 dimGrid2(1,1);
 dim3 dimBlock2(1,1);

 checkCudaErrors(hipEventRecord(start));              
 PowerKernal1<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, iterations);  
 checkCudaErrors(hipEventRecord(stop));               
 
 checkCudaErrors(hipEventSynchronize(stop));           
 checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));  
 printf("execution time = %.2f s\n", elapsedTime/1000);  
 getLastCudaError("kernel launch failure");              
 hipDeviceSynchronize();

/*CUT_SAFE_CALL(cutCreateTimer(&my_timer)); 
TaskHandle taskhandle = LaunchDAQ();
CUT_SAFE_CALL(cutStartTimer(my_timer)); 
printf("execution time = %f\n", cutGetTimerValue(my_timer));



PowerKernal1<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
CUDA_SAFE_CALL( hipDeviceSynchronize() );
printf("execution time = %f\n", cutGetTimerValue(my_timer));


getLastCudaError("kernel launch failure");
CUDA_SAFE_CALL( hipDeviceSynchronize() );
CUT_SAFE_CALL(cutStopTimer(my_timer));
TurnOffDAQ(taskhandle, cutGetTimerValue(my_timer));
printf("execution time = %f\n", cutGetTimerValue(my_timer));
CUT_SAFE_CALL(cutDeleteTimer(my_timer)); 

#ifdef _DEBUG
 checkCudaErrors( hipDeviceSynchronize() );
#endif*/

 // Copy result from device memory to host memory
 // h_C contains the result in host memory
 checkCudaErrors( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
 
 checkCudaErrors(hipEventDestroy(start));
 checkCudaErrors(hipEventDestroy(stop));
 CleanupResources();

 return 0;
}

void CleanupResources(void)
{
  // Free device memory
  if (d_A)
	hipFree(d_A);
  if (d_B)
	hipFree(d_B);
  if (d_C)
	hipFree(d_C);

  // Free host memory
  if (h_A)
	free(h_A);
  if (h_B)
	free(h_B);
  if (h_C)
	free(h_C);

}

// Allocates an array with random float entries.
void RandomInit(double* data, int n)
{
  for (int i = 0; i < n; ++i){ 
	data[i] = rand() / RAND_MAX;
  }
}






