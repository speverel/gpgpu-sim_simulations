#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include <stdlib.h>


// includes CUDA
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256
#define NUM_OF_BLOCKS 80
#define F 20
#define ITERATIONS (unsigned)( 10000 )
#define max_tid THREADS_PER_BLOCK*NUM_OF_BLOCKS


// Variables
int* h_A;
int* h_B;
int* h_C;
int* d_A;
int* d_B;
int* d_C;


// Functions
void CleanupResources(void);
void RandomInit(int*, int);


////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line ){
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line ){
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions




// Device code
__global__ void PowerKernal(int* A, int* C, int iterations){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation


    int m_sum=0;

    for (unsigned j=0; j<iterations; j++){
    	for(unsigned k=0; k<ITERATIONS; ++k){
		m_sum=A[((unsigned)(tid*F)+(unsigned)(k*max_tid*F))];
    	}
	m_sum+=j;
    }
    C[tid]=m_sum;
   __syncthreads();
}


// Host code

int main(int argc, char** argv) 
{
	int iterations;
	if (argc != 2){
		fprintf(stderr,"usage: %s #iterations\n",argv[0]);
		exit(1);
	}
	else{
		iterations = atoi(argv[1]);
	}

	printf("Power Microbenchmark with %d iterations\n",iterations);
	unsigned N =((unsigned)(max_tid*F)+(unsigned)(ITERATIONS*max_tid*F));
	size_t size = N * sizeof(int);

	// Allocate input vectors h_A and h_B in host memory
	h_A = (int*)malloc(size);
	if (h_A == 0) CleanupResources();
	//h_B = (float*)malloc(size);
	//if (h_B == 0) CleanupResources();
	h_C = (int*)malloc(size);
	if (h_C == 0) CleanupResources();

	// Initialize input vectors
	RandomInit(h_A, N);
	//RandomInit(h_B, N);

	// Allocate vectors in device memory
	checkCudaErrors( hipMalloc((void**)&d_A, size) );
	//checkCudaErrors( hipMalloc((void**)&d_B, size) );
	checkCudaErrors( hipMalloc((void**)&d_C, size) );

	// Copy vectors from host memory to device memory
	checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );


	hipEvent_t start, stop;
	float elapsedTime = 0;
	checkCudaErrors(hipEventCreate(&start));
	checkCudaErrors(hipEventCreate(&stop));
	dim3 dimGrid(NUM_OF_BLOCKS,1);
	dim3 dimBlock(THREADS_PER_BLOCK,1);

	checkCudaErrors(hipEventRecord(start));
	PowerKernal<<<dimGrid,dimBlock>>>(d_A, d_C, iterations);

	checkCudaErrors(hipEventRecord(stop));

	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("gpu execution time = %.2f s\n", elapsedTime/1000);

	getLastCudaError("kernel launch failure");
	hipDeviceSynchronize();

	 // Copy result from device memory to host memory
	 // h_C contains the result in host memory
	checkCudaErrors( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );

	checkCudaErrors(hipEventDestroy(start));
	checkCudaErrors(hipEventDestroy(stop));
	CleanupResources();

	return 0;
}

void CleanupResources(void){
  // Free device memory
  if (d_A)
	hipFree(d_A);
  //if (d_B)
//	hipFree(d_B);
  if (d_C)
	hipFree(d_C);

  // Free host memory
  if (h_A)
	free(h_A);
 // if (h_B)
//	free(h_B);
  if (h_C)
	free(h_C);

}

// Allocates an array with random float entries.
void RandomInit(int* data, int n){
  for (int i = 0; i < n; ++i)
	data[i] = (int)(rand() / RAND_MAX);
}
