#include "hip/hip_runtime.h"
#include <stdio.h>   
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 1024
#define DATA_TYPE float
#define LINE_SIZE   128
#define SETS    4
#define ASSOC   24
#define SIMD_WIDTH  32


// Variables
int no_of_nodes;
int edge_list_size;
FILE *fp;

//Structure to hold a node information
struct Node
{
  int starting;
  int no_of_edges;
};


// GPU error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true){
  if (code != hipSuccess) {
    fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}


////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line ){
  if(hipSuccess != err){
  fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
   exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line ){
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
  fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
  exit(-1);
  }
}
// end of CUDA Helper Functions




// Device code


texture<float,1,hipReadModeElementType> texmem1;
texture<float,1,hipReadModeElementType> texmem2;
texture<float,1,hipReadModeElementType> texmem3;
texture<float,1,hipReadModeElementType> texmem4;
texture<float,1,hipReadModeElementType> texmem5;
texture<float,1,hipReadModeElementType> texmem6;
texture<float,1,hipReadModeElementType> texmem7;
texture<float,1,hipReadModeElementType> texmem9;
texture<float,1,hipReadModeElementType> texmem8;

__global__ void power_microbench(float *data1, float *data2, uint32_t *data3, uint32_t *data4, float *res, int div, uint64_t iterations, float* out, unsigned size) {

  int gid = blockIdx.x*blockDim.x + threadIdx.x;
  register float s1 = data1[gid];
  register float s2 = data2[gid];
  register uint32_t s3 = data3[gid];
  register uint32_t s4 = data4[gid];
  register float result = 0;
  register float Value1=0;
  register uint32_t Value2=0;
  register float Value3=0;
  // synchronize all threads
  asm volatile ("bar.sync 0;");

  if(gid < size){
    if((gid%32)<div){
    //ROI
      #pragma unroll 1000
      for (uint64_t j=0 ; j<iterations ; ++j) {
        asm volatile ("{\t\n"
            "add.f32 %0, %1, %0;\n\t"
            "add.u32 %2, %3, %2;\n\t"
            "add.u32 %2, %3, %2;\n\t"
            // "add.u32 %2, %2, %0;\n\t"
            // "mul.lo.u32 %1, %0, %2;\n\t"
            "fma.rn.f32 %1, %1, %1 , %0;\n\t"
            "mad.lo.u32 %3, %3, %3 , %2;\n\t"
            "}" : "+f"(Value1),"+f"(s1),"+r"(s3),"+r"(Value2)
        );
        Value3 += tex1Dfetch(texmem1,Value2%gid);
        // Value3 += tex1Dfetch(texmem2,Value2%gid);
        // Value3 += tex1Dfetch(texmem3,Value2%gid);
        // Value3 += tex1Dfetch(texmem4,Value2%gid);
        // Value3 += tex1Dfetch(texmem5,Value2%gid);
        // Value3 += tex1Dfetch(texmem6,Value2%gid);
        // Value3 += tex1Dfetch(texmem7,Value2%gid);
        // Value3 += tex1Dfetch(texmem8,Value2%gid);
        // Value3 += tex1Dfetch(texmem9,Value2%gid);
      }
    }
  }

  // synchronize all threads
  asm volatile("bar.sync 0;");

  // write data back to memory
  res[gid] = Value1 + (float)Value2;
  out[gid] = Value3;
}

int main(int argc, char** argv){
  uint64_t iterations;
  int blocks;
  int div;
  if (argc != 4){
    fprintf(stderr,"usage: %s #iterations #cores #ActiveThreadsperWarp\n",argv[0]);
    exit(1);
  }
  else {
    iterations = atoll(argv[1]);
    blocks = atoi(argv[2]);
    div = atoi(argv[3]);
  }
 
 printf("Power Microbenchmarks with iterations %lu\n",iterations);


 
 

  unsigned num_blocks = blocks;
  int texmem_size = THREADS_PER_BLOCK*num_blocks;
  dim3  grid( num_blocks, 1, 1);
  dim3  threads( THREADS_PER_BLOCK, 1, 1);
  int total_threads = THREADS_PER_BLOCK*num_blocks;

 float *host_texture1 = (float*) malloc(texmem_size*sizeof(float));
  for (int i=0; i< texmem_size; i++) {
    host_texture1[i] = i;
  }
  float *device_texture1;
  float *device_texture2;
  float *device_texture3;
  float *device_texture4;
  float *device_texture5;
  float *device_texture6;
  float *device_texture7;
  float *device_texture8;
  float *device_texture9;

  float *host_out = (float*) malloc(texmem_size*sizeof(float)*10);
  float *device_out;

  hipMalloc((void**) &device_texture1, texmem_size*sizeof(float));
  hipMalloc((void**) &device_texture2, texmem_size*sizeof(float));
  hipMalloc((void**) &device_texture3, texmem_size*sizeof(float));
  hipMalloc((void**) &device_texture4, texmem_size*sizeof(float));
  hipMalloc((void**) &device_texture5, texmem_size*sizeof(float));
  hipMalloc((void**) &device_texture6, texmem_size*sizeof(float));
  hipMalloc((void**) &device_texture7, texmem_size*sizeof(float));
  hipMalloc((void**) &device_texture8, texmem_size*sizeof(float));
  hipMalloc((void**) &device_texture9, texmem_size*sizeof(float));

  hipMalloc((void**) &device_out, texmem_size*10);

  hipMemcpy(device_texture1, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_texture2, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_texture3, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_texture4, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_texture5, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_texture6, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_texture7, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_texture8, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_texture9, host_texture1, texmem_size*sizeof(float), hipMemcpyHostToDevice);

  hipBindTexture(0, texmem1, device_texture1, texmem_size*sizeof(float));
  hipBindTexture(0, texmem2, device_texture2, texmem_size*sizeof(float));
  hipBindTexture(0, texmem3, device_texture3, texmem_size*sizeof(float));
  hipBindTexture(0, texmem4, device_texture4, texmem_size*sizeof(float));
  hipBindTexture(0, texmem5, device_texture5, texmem_size*sizeof(float));
  hipBindTexture(0, texmem6, device_texture6, texmem_size*sizeof(float));
  hipBindTexture(0, texmem7, device_texture7, texmem_size*sizeof(float));
  hipBindTexture(0, texmem8, device_texture8, texmem_size*sizeof(float));
  hipBindTexture(0, texmem9, device_texture9, texmem_size*sizeof(float));

DATA_TYPE *data1 = (DATA_TYPE*) malloc(total_threads*sizeof(DATA_TYPE));
DATA_TYPE *data2 = (DATA_TYPE*) malloc(total_threads*sizeof(DATA_TYPE));


uint32_t *data3 = (uint32_t*) malloc(total_threads*sizeof(uint32_t));
uint32_t *data4 = (uint32_t*) malloc(total_threads*sizeof(uint32_t));

DATA_TYPE *res = (DATA_TYPE*) malloc(total_threads*sizeof(DATA_TYPE));

DATA_TYPE *data1_g;
DATA_TYPE *data2_g;

uint32_t *data3_g;
uint32_t *data4_g;

DATA_TYPE *res_g;

for (uint32_t i=0; i<total_threads; i++) {
  srand((unsigned)time(0));  
  data1[i] = (DATA_TYPE) rand() / RAND_MAX;
  srand((unsigned)time(0));
  data2[i] = (DATA_TYPE) rand() / RAND_MAX;
  srand((unsigned)time(0));  
  data3[i] = (uint32_t) rand() / RAND_MAX;
  srand((unsigned)time(0));
  data4[i] = (uint32_t) rand() / RAND_MAX;
}

gpuErrchk( hipMalloc(&data1_g, total_threads*sizeof(DATA_TYPE)) );
gpuErrchk( hipMalloc(&data2_g, total_threads*sizeof(DATA_TYPE)) );

gpuErrchk( hipMalloc(&data3_g, total_threads*sizeof(uint32_t)) );
gpuErrchk( hipMalloc(&data4_g, total_threads*sizeof(uint32_t)) );

gpuErrchk( hipMalloc(&res_g, total_threads*sizeof(DATA_TYPE)) );

gpuErrchk( hipMemcpy(data1_g, data1, total_threads*sizeof(DATA_TYPE), hipMemcpyHostToDevice) );
gpuErrchk( hipMemcpy(data2_g, data2, total_threads*sizeof(DATA_TYPE), hipMemcpyHostToDevice) );

gpuErrchk( hipMemcpy(data3_g, data3, total_threads*sizeof(uint32_t), hipMemcpyHostToDevice) );
gpuErrchk( hipMemcpy(data4_g, data4, total_threads*sizeof(uint32_t), hipMemcpyHostToDevice) );


power_microbench<<<grid,threads,0>>>(data1_g, data2_g, data3_g, data4_g, res_g, div, iterations, device_out, texmem_size);
gpuErrchk( hipPeekAtLastError() );


gpuErrchk( hipMemcpy(res, res_g, total_threads*sizeof(DATA_TYPE), hipMemcpyDeviceToHost) );
hipMemcpy(host_out, device_out, texmem_size*sizeof(float), hipMemcpyDeviceToHost);


hipFree(data1_g);
hipFree(data2_g);
hipFree(data3_g);
hipFree(data4_g);
hipFree(res_g);
  hipUnbindTexture(texmem1);
  hipUnbindTexture(texmem2);
  hipUnbindTexture(texmem3);
  hipUnbindTexture(texmem4);
  hipUnbindTexture(texmem5);
  hipUnbindTexture(texmem6);
  hipUnbindTexture(texmem7);
  hipUnbindTexture(texmem8);
  hipUnbindTexture(texmem9);
free(data1);
free(data2);
free(data3);
free(data4);
free(res);

  return 0;
} 
