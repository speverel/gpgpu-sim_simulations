#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
//#include <cutil.h>
// Includes
#include <stdio.h>
#include<hip/hip_runtime.h>
// includes, project
//#include "../include/sdkHelper.h"  // helper for shared functions common to CUDA SDK samples
//#include <shrQATest.h>
//#include <shrUtils.h>

// includes CUDA
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 1024

// Variables
uint32_t* h_A;
uint32_t* h_B;
uint32_t* h_C;
uint32_t* d_A;
uint32_t* d_B;
uint32_t* d_C;
bool noprompt = false;
unsigned int my_timer;

// Functions
void CleanupResources(void);
void RandomInit(uint32_t*, int);
void ParseArguments(int, char**);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
	fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
	 exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
	fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
	exit(-1);
  }
}

// end of CUDA Helper Functions
__global__ void PowerKernal2(const uint32_t* A, const uint32_t* B, uint32_t* C, int N, uint64_t iterations, int div)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    uint32_t Value1=1;
    uint32_t Value2=A[i];
    uint32_t Value3=B[i];
    uint32_t Value;
    uint32_t I1=A[i];
    uint32_t I2=B[i];
    
    // Excessive Addition access
//    if(((i%32)<=31))
if((i%2)==0){    
    #pragma unroll 1000
    for(uint64_t k=0; k<iterations;k++) {
	Value1=I1*A[i];
	Value3=I2*B[i];
	Value1*=Value2;
	Value1*=Value2;
	Value2=Value3*Value1;
	Value1=Value2*Value3;
    }
}
    __syncthreads();

    Value=Value1;
    C[i]=Value*Value2;

}

int main(int argc, char** argv)
{
  uint64_t iterations;
  unsigned blocks;
  int div;
  if (argc != 4){
	  fprintf(stderr,"usage: %s #iterations #cores\n",argv[0]);
	  exit(1);
  }
  else {
    iterations = atoll(argv[1]);
    blocks = atoi(argv[2]);
    div = atoi(argv[3]);
  }

 printf("Power Microbenchmarks with %llu iterations \n", iterations);
 int N = THREADS_PER_BLOCK*blocks;
 size_t size = N * sizeof(uint32_t);
 // Allocate input vectors h_A and h_B in host memory
 h_A = (uint32_t*)malloc(size);
 if (h_A == 0) CleanupResources();
 h_B = (uint32_t*)malloc(size);
 if (h_B == 0) CleanupResources();
 h_C = (uint32_t*)malloc(size);
 if (h_C == 0) CleanupResources();

 // Initialize input vectors
 RandomInit(h_A, N);
 RandomInit(h_B, N);

 // Allocate vectors in device memory
printf("before\n");
 checkCudaErrors( hipMalloc((void**)&d_A, size) );
 checkCudaErrors( hipMalloc((void**)&d_B, size) );
 checkCudaErrors( hipMalloc((void**)&d_C, size) );
printf("after\n");

 // Copy vectors from host memory to device memory
 checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
 checkCudaErrors( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );

 //VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
 dim3 dimGrid(blocks,1);
 dim3 dimBlock(THREADS_PER_BLOCK,1);
 dim3 dimGrid2(1,1);
 dim3 dimBlock2(1,1);



PowerKernal2<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N, iterations,div);
hipDeviceSynchronize();


getLastCudaError("kernel launch failure");
hipDeviceSynchronize();

#ifdef _DEBUG
 checkCudaErrors( hipDeviceSynchronize() );
#endif

 // Copy result from device memory to host memory
 // h_C contains the result in host memory
 checkCudaErrors( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
 
 CleanupResources();

 return 0;
}

void CleanupResources(void)
{
  // Free device memory
  if (d_A)
	hipFree(d_A);
  if (d_B)
	hipFree(d_B);
  if (d_C)
	hipFree(d_C);

  // Free host memory
  if (h_A)
	free(h_A);
  if (h_B)
	free(h_B);
  if (h_C)
	free(h_C);

}

// Allocates an array with random uint32_t entries.
void RandomInit(uint32_t* data, int n)
{
  for (int i = 0; i < n; ++i){ 
	data[i] = rand() / RAND_MAX;
  }
}






