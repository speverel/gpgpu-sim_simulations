#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
//#include <cutil.h>
// Includes
//#include <stdio.h>

// includes, project
//#include "../include/sdkHelper.h"  // helper for shared functions common to CUDA SDK samples
//#include <shrQATest.h>
//#include <shrUtils.h>

// includes CUDA
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256
#define NUM_OF_BLOCKS 640
//#define ITERATIONS 40
//#include "../include/ContAcq-IntClk.h"

// Variables
float* h_A;
float* h_B;
float* h_C;
float* d_A;
float* d_B;
float* d_C;
//bool noprompt = false;
//unsigned int my_timer;

// Functions
void CleanupResources(void);
void RandomInit(float*, int);
//void ParseArguments(int, char**);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
  fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
   exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
  fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
  exit(-1);
  }
}

// end of CUDA Helper Functions
__global__ void PowerKernal2(const float* A, const float* B, float* C, int iterations)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    float def1, def2, def3, def4, def5, def6;
    float use1, use2, use3, use4, use5, use6 = 0;
    float I1=A[i];
    float I2=B[i];
    float Value;
    
  #pragma unroll 100
    // Excessive Addition access
    for(unsigned k=0; k<iterations;k++) {
      def1= __fmaf_rn(I1,I2,use1);
      def2= __fmaf_rn(I1,I2,use2);
      def3= __fmaf_rn(I1,I2,use3);
      def4= __fmaf_rn(I1,I2,use4);
      def5= __fmaf_rn(I1,I2,use5);
      def6= __fmaf_rn(I1,I2,use6);

      use1= __fmaf_rn(def3,def1,def2); 
      use2= __fmaf_rn(def1,def4,def2); 
      use3= __fmaf_rn(def1,def2,def3);
      use4= __fmaf_rn(def6,def4,def5); 
      use5= __fmaf_rn(def4,def2,def5); 
      use6= __fmaf_rn(def4,def5,def6);
    }
    __syncthreads();

    Value=(use1+use2)* (use3 + use4) * (use5 + use6);
    C[i]=Value;
    __syncthreads();

}

int main(int argc, char** argv)
{
 int iterations;
 if(argc!=2) {
   fprintf(stderr,"usage: %s #iterations\n",argv[0]);
   exit(1);
 }
 else {
   iterations = atoi(argv[1]);
 }

 printf("Power Microbenchmarks with iterations %d\n",iterations);
 int N = THREADS_PER_BLOCK*NUM_OF_BLOCKS;
 size_t size = N * sizeof(float);
 // Allocate input vectors h_A and h_B in host memory
 h_A = (float*)malloc(size);
 if (h_A == 0) CleanupResources();
 h_B = (float*)malloc(size);
 if (h_B == 0) CleanupResources();
 h_C = (float*)malloc(size);
 if (h_C == 0) CleanupResources();

 // Initialize input vectors
 RandomInit(h_A, N);
 RandomInit(h_B, N);

 // Allocate vectors in device memory
printf("before\n");
 checkCudaErrors( hipMalloc((void**)&d_A, size) );
 checkCudaErrors( hipMalloc((void**)&d_B, size) );
 checkCudaErrors( hipMalloc((void**)&d_C, size) );
printf("after\n");

 hipEvent_t start, stop;                   
 float elapsedTime = 0;                     
 checkCudaErrors(hipEventCreate(&start));  
 checkCudaErrors(hipEventCreate(&stop));

 // Copy vectors from host memory to device memory
 checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
 checkCudaErrors( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );

 //VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
 dim3 dimGrid(NUM_OF_BLOCKS,1);
 dim3 dimBlock(THREADS_PER_BLOCK,1);
 dim3 dimGrid2(1,1);
 dim3 dimBlock2(1,1);

 checkCudaErrors(hipEventRecord(start));              
 PowerKernal2<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, iterations);  
 checkCudaErrors(hipEventRecord(stop));               
 
 checkCudaErrors(hipEventSynchronize(stop));           
 checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));  
 printf("execution time = %.2f s\n", elapsedTime/1000);  
 getLastCudaError("kernel launch failure");              
 hipDeviceSynchronize();

/*CUT_SAFE_CALL(cutCreateTimer(&my_timer)); 
TaskHandle taskhandle = LaunchDAQ();
CUT_SAFE_CALL(cutStartTimer(my_timer)); 
printf("execution time = %f\n", cutGetTimerValue(my_timer));
PowerKernal2<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, N);
CUDA_SAFE_CALL( hipDeviceSynchronize() );
printf("execution time = %f\n", cutGetTimerValue(my_timer));


getLastCudaError("kernel launch failure");
CUDA_SAFE_CALL( hipDeviceSynchronize() );
CUT_SAFE_CALL(cutStopTimer(my_timer));
TurnOffDAQ(taskhandle, cutGetTimerValue(my_timer));
printf("execution time = %f\n", cutGetTimerValue(my_timer));
CUT_SAFE_CALL(cutDeleteTimer(my_timer)); 

#ifdef _DEBUG
 checkCudaErrors( hipDeviceSynchronize() );
#endif*/

 // Copy result from device memory to host memory
 // h_C contains the result in host memory
 checkCudaErrors( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
 checkCudaErrors(hipEventDestroy(start));
 checkCudaErrors(hipEventDestroy(stop));
 CleanupResources();

 return 0;
}

void CleanupResources(void)
{
  // Free device memory
  if (d_A)
  hipFree(d_A);
  if (d_B)
  hipFree(d_B);
  if (d_C)
  hipFree(d_C);

  // Free host memory
  if (h_A)
  free(h_A);
  if (h_B)
  free(h_B);
  if (h_C)
  free(h_C);

}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
  for (int i = 0; i < n; ++i){ 
  data[i] = rand() / RAND_MAX;
  }
}






