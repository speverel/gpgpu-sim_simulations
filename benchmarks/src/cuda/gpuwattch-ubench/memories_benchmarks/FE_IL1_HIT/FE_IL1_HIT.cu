#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include <stdlib.h>


// includes CUDA
#include <hip/hip_runtime.h>

//includes project
#include<repeat.h>

#define THREADS_PER_BLOCK 256
#define NUM_OF_BLOCKS 640


// Variables
unsigned* h_A;
unsigned* h_C;
unsigned* d_A;
unsigned* d_C;


// Functions
void CleanupResources(void);
void RandomInit(unsigned*, int);


////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
     exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
    fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
    exit(-1);
  }
}

// end of CUDA Helper Functions




// Device code
__global__ void PowerKernal(const unsigned* A,unsigned* C, int iterations)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    //Do Some Computation
    unsigned I1=A[i];
    #pragma unroll 1
    //Excessive Logical Unit access
    for(unsigned k=0; k<iterations;k++) {
    // BLOCK-0 (For instruction size of 16 bytes)
    	__asm volatile (	
    	"B0: bra.uni B1;\n\t"
    	repeat767("add.u32   %0, %1, 1;\n\t")

    	"B1: bra.uni B2;\n\t"
    	repeat767("add.u32   %0, %1, 1;\n\t")
    			
    	"B2: bra.uni B3;\n\t"
    	repeat767("add.u32   %0, %1, 1;\n\t")

    	"B3: bra.uni B4;\n\t"
    	repeat767("add.u32   %0, %1, 1;\n\t")

    	"B4: bra.uni B5;\n\t"
    	repeat767("add.u32   %0, %1, 1;\n\t")

    	"B5: bra.uni B6;\n\t"
    	repeat767("add.u32   %0, %1, 1;\n\t")
    	
    	"B6: bra.uni LOOP;\n\t"
    	repeat767("add.u32   %0, %1, 1;\n\t")
        
    	"LOOP:"
    	: "=r"(I1) : "r"(I1));
    	
    }
    C[i]=I1;
    __syncthreads();

}

// Host code

int main(int argc, char** argv) 
{

  int iterations;
  if (argc != 2){
    fprintf(stderr,"usage: %s #iterations\n",argv[0]);
    exit(1);
  }
  else{
    iterations = atoi(argv[1]);
  }

  printf("Power Microbenchmark with %d iterations\n",iterations);
 int N = THREADS_PER_BLOCK*NUM_OF_BLOCKS;
 size_t size = N * sizeof(unsigned);
 // Allocate input vectors h_A and h_B in host memory
 h_A = (unsigned*)malloc(size);
 if (h_A == 0) CleanupResources();
 h_C = (unsigned*)malloc(size);
 if (h_C == 0) CleanupResources();

 // Initialize input vectors
 RandomInit(h_A, N);

 // Allocate vectors in device memory
 checkCudaErrors( hipMalloc((void**)&d_A, size) );
 checkCudaErrors( hipMalloc((void**)&d_C, size) );

 // Copy vectors from host memory to device memory
 checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );

 hipEvent_t start, stop;
 float elapsedTime = 0;
 checkCudaErrors(hipEventCreate(&start));
 checkCudaErrors(hipEventCreate(&stop));
 dim3 dimGrid(NUM_OF_BLOCKS,1);
 dim3 dimBlock(THREADS_PER_BLOCK,1);

 checkCudaErrors(hipEventRecord(start));
 PowerKernal<<<dimGrid,dimBlock>>>(d_A,d_C, iterations);
 checkCudaErrors(hipEventRecord(stop));

 checkCudaErrors(hipEventSynchronize(stop));
 checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
 printf("gpu execution time = %.2f s\n", elapsedTime/1000);

 getLastCudaError("kernel launch failure");
 hipDeviceSynchronize();

 // Copy result from device memory to host memory
 // h_C contains the result in host memory
 checkCudaErrors(hipEventDestroy(start));
 checkCudaErrors(hipEventDestroy(stop));
 checkCudaErrors( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
 
 CleanupResources();

 return 0;
}

void CleanupResources(void)
{
  // Free device memory
  if (d_A)
    hipFree(d_A);
  if (d_C)
    hipFree(d_C);

  // Free host memory
  if (h_A)
    free(h_A);
  if (h_C)
    free(h_C);

}

// Allocates an array with random float entries.
void RandomInit(unsigned* data, int n)
{
  for (int i = 0; i < n; ++i){
    srand((unsigned)time(0));  
    data[i] = rand() / RAND_MAX;
  }
}






