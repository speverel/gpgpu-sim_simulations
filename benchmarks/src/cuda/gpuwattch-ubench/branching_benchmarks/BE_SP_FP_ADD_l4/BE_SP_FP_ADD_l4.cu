#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include <stdlib.h>
// includes from project


// includes from CUDA
#include <hip/hip_runtime.h>
//#include <hip/hip_vector_types.h>

#define THREADS_PER_BLOCK 256
#define NUM_OF_BLOCKS 640


// Variables
float* h_A;
float* h_B;
float* h_C;
float* d_A;
float* d_B;
float* d_C;

// Functions
void CleanupResources(void);
void RandomInit(float*, int);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
  fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
   exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
  fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
  exit(-1);
  }
}
// end of CUDA Helper Functions

__global__ void PowerKernal2(const float* A, const float* B, float* C, int iterations)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  //Do Some Computation
  float Value1 = 0;
  float Value2 = 0;
  float Value3;
  float Value;
  float I1=A[i];
  float I2=B[i];

  // Excessive Addition access
  if((i%32)<=3){
    #pragma unroll 100
    for(unsigned k=0; k<iterations;k++) {
      Value1=I1+I2;
      Value3=I1-I2;
      Value1+=Value2;
      Value1+=Value2;
      Value2=Value3-Value1;
      Value1=Value2+Value3;
    }
  }
  __syncthreads();

  Value=Value1;
  C[i]=Value+Value2;
}

int main(int argc, char** argv) 
{

  int iterations;
  if (argc != 2){
  fprintf(stderr,"usage: %s #iterations\n",argv[0]);
  exit(1);
  }
  else{
    iterations = atoi(argv[1]);
  }

 printf("Power Microbenchmark with %d iterations\n",iterations);
 int N = THREADS_PER_BLOCK*NUM_OF_BLOCKS;
 size_t size = N * sizeof(float);
 // Allocate input vectors h_A and h_B in host memory
 h_A = (float*)malloc(size);
 if (h_A == 0) CleanupResources();
 h_B = (float*)malloc(size);
 if (h_B == 0) CleanupResources();
 h_C = (float*)malloc(size);
 if (h_C == 0) CleanupResources();



 // Initialize input vectors
 RandomInit(h_A, N);
 RandomInit(h_B, N);

 // Allocate vectors in device memory
 printf("before\n");
 checkCudaErrors( hipMalloc((void**)&d_A, size) );
 checkCudaErrors( hipMalloc((void**)&d_B, size) );
 checkCudaErrors( hipMalloc((void**)&d_C, size) );
 printf("after\n");

 hipEvent_t start, stop;
 float elapsedTime = 0;
 checkCudaErrors(hipEventCreate(&start));
 checkCudaErrors(hipEventCreate(&stop));

 // Copy vectors from host memory to device memory
 checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );
 checkCudaErrors( hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice) );


 dim3 dimGrid(NUM_OF_BLOCKS,1);
 dim3 dimBlock(THREADS_PER_BLOCK,1);
 dim3 dimGrid2(1,1);
 dim3 dimBlock2(1,1);

 checkCudaErrors(hipEventRecord(start));
 PowerKernal2<<<dimGrid,dimBlock>>>(d_A, d_B, d_C, iterations);
 checkCudaErrors(hipEventRecord(stop));

 checkCudaErrors(hipEventSynchronize(stop));
 checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
 printf("gpu execution time = %.2f s\n", elapsedTime/1000);

 getLastCudaError("kernel launch failure");
 hipDeviceSynchronize();

 // Copy result from device memory to host memory
 // h_C contains the result in host memory
 checkCudaErrors( hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost) );
 
 checkCudaErrors(hipEventDestroy(start));
 checkCudaErrors(hipEventDestroy(stop));
 CleanupResources();

 return 0;
}

void CleanupResources(void)
{
  // Free device memory
  if (d_A)
  hipFree(d_A);
  if (d_B)
  hipFree(d_B);
  if (d_C)
  hipFree(d_C);

  // Free host memory
  if (h_A)
  free(h_A);
  if (h_B)
  free(h_B);
  if (h_C)
  free(h_C);

}

// Allocates an array with random float entries.
void RandomInit(float* data, int n)
{
  for (int i = 0; i < n; ++i){ 
  data[i] = rand() / RAND_MAX;
  }
}