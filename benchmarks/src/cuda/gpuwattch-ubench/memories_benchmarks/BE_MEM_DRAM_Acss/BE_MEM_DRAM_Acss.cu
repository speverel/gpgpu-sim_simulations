#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
//#include <cutil.h>
//#include <mgp.h>
// Includes
//#include <stdio.h>
//#include "../include/ContAcq-IntClk.h"

// includes, project
//#include "../include/sdkHelper.h"  // helper for shared functions common to CUDA SDK samples
//#include <shrQATest.h>
//#include <shrUtils.h>

// includes CUDA
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 256
#define NUM_OF_BLOCKS 640
#define NUM_SM 80
#define LINE_SIZE   64
#define SETS    4096
#define ASSOC   18
#define NUMTHREADS THREADS_PER_BLOCK*NUM_OF_BLOCKS
#define ITERATIONS 100

// Variables
unsigned* h_A;
unsigned* h_B;
unsigned* h_C;
unsigned* d_A;
unsigned* d_B;
unsigned* d_C;
//bool noprompt = false;
//unsigned int my_timer;

// Functions
void CleanupResources(void);
void RandomInit(unsigned*, unsigned long);
//void ParseArguments(int, char**);

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
  if(hipSuccess != err){
  fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
   exit(-1);
  }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
  hipError_t err = hipGetLastError();
  if (hipSuccess != err){
  fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",file, line, errorMessage, (int)err, hipGetErrorString( err ) );
  exit(-1);
  }
}

// end of CUDA Helper Functions



__global__ void PowerKernal2( unsigned* A, unsigned* B, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    unsigned load_value;
  //unsigned sum_value = 0;
  // unsigned * loadAddr = &A[i];
  //   unsigned * storeAddr = &B[i];
    unsigned size_l2 = (LINE_SIZE*ASSOC*SETS);
    unsigned stride = size_l2/sizeof(unsigned) -1;
    #pragma unroll 100
    for(unsigned iterations=0; iterations<N;iterations++) {    
        unsigned * loadAddr = &A[i];
        unsigned * storeAddr = &B[i];
        #pragma unroll 10
       for(unsigned k =0; k<ITERATIONS; k++){
        __asm volatile(
          "ld.global.cv.u32 %0, [%1];" 
          : "=r"(load_value) : "l"((unsigned long)(loadAddr))
        );
        //__asm volatile("add.u32 %0, %0, %1;" : "+r"(sum_value) : "r"(load_value));
        __asm volatile(
          "st.global.wt.u32 [%0], %1;"
          : : "l"((unsigned long)(storeAddr)), "r"(load_value) 
        );
        loadAddr = loadAddr + stride;
        storeAddr = storeAddr + stride;
       }
    }
    //B[i] = sum_value;
    __syncthreads();

}


int main(int argc, char** argv)
{
 int iterations;
 if(argc!=2) {
   fprintf(stderr,"usage: %s #iterations\n",argv[0]);
   exit(1);
 }
 else {
   iterations = atoi(argv[1]);
 }
 
 printf("Power Microbenchmarks with iterations %d\n",iterations);

 unsigned long size_l2 = (LINE_SIZE*ASSOC*SETS);
 unsigned long N = size_l2*ITERATIONS;
 size_t size = N * sizeof(unsigned);
 // Allocate input vectors h_A and h_B in host memory
 h_A = (unsigned*)malloc(size);
 if (h_A == 0) CleanupResources();
 h_B = (unsigned*)malloc(size);
 if (h_B == 0) CleanupResources();


 // Initialize input vectors
 RandomInit(h_A, N);


 // Allocate vectors in device memory
 checkCudaErrors( hipMalloc((void**)&d_A, size) );
 checkCudaErrors( hipMalloc((void**)&d_B, size) );


 // Copy vector from host memory to device memory
 checkCudaErrors( hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice) );


 hipEvent_t start, stop;                   
 float elapsedTime = 0;                     
 checkCudaErrors(hipEventCreate(&start));  
 checkCudaErrors(hipEventCreate(&stop));

 //VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
 dim3 dimGrid(NUM_OF_BLOCKS,1);
 dim3 dimBlock(THREADS_PER_BLOCK,1);


 checkCudaErrors(hipEventRecord(start));              
 PowerKernal2<<<dimGrid,dimBlock>>>(d_A, d_B,iterations);  
 checkCudaErrors(hipEventRecord(stop));               
 
 checkCudaErrors(hipEventSynchronize(stop));           
 checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));  
 printf("execution time = %.2f s\n", elapsedTime/1000);  
 getLastCudaError("kernel launch failure");              
 hipDeviceSynchronize(); 

 // Copy result from device memory to host memory
 // h_B contains the result in host memory
 checkCudaErrors( hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost) );
  checkCudaErrors(hipEventDestroy(start));
 checkCudaErrors(hipEventDestroy(stop));
 CleanupResources();

 return 0;
}

void CleanupResources(void)
{
  // Free device memory
  if (d_A)
  hipFree(d_A);
  if (d_B)
  hipFree(d_B);

  // Free host memory
  if (h_A)
  free(h_A);
  if (h_B)
  free(h_B);

}

// Allocates an array with random float entries.
void RandomInit(unsigned* data, unsigned long n)
{
  for (unsigned long i = 0; i < n; ++i){
  //srand((unsigned)time(0));  
  data[i] = i;
  }
}